
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount); // Get the number of devices
    printf("Number of CUDA devices: %d\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i); // Get properties of device i
        printf("\nDevice %d: %s\n", i, prop.name);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total Global Memory: %zu MB\n", prop.totalGlobalMem / (1024 * 1024));
        printf("  Shared Memory per Block: %zu KB\n", prop.sharedMemPerBlock / 1024);
        printf("  Registers per Block: %d\n", prop.regsPerBlock);
        printf("  Warp Size: %d\n", prop.warpSize);
        printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max Threads Dimensions: [%d, %d, %d]\n", 
               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max Grid Dimensions: [%d, %d, %d]\n", 
               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
    return 0;
}
