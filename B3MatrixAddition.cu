
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixAdd(int *a, int *b, int *c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread index
    if (idx < size) c[idx] = a[idx] + b[idx];
}

int main() {
    int rows = 2, cols = 3, size = rows * cols;
    int a[] = {1, 2, 3, 4, 5, 6}, b[] = {10, 20, 30, 40, 50, 60}, c[size];

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    matrixAdd<<<1, size>>>(d_a, d_b, d_c, size); // Single block, one thread per element

    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) printf("%d ", c[i]);
    printf("\n");

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
