
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 3 // Size of the matrices (N x N)

// Kernel function for matrix multiplication
__global__ void matrixMultiply(int *a, int *b, int *c, int n) {
    int row = threadIdx.y; // Row index of the thread
    int col = threadIdx.x; // Column index of the thread
    int sum = 0;

    if (row < n && col < n) {
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    int a[N][N] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
    int b[N][N] = {{9, 8, 7}, {6, 5, 4}, {3, 2, 1}};
    int c[N][N] = {0};

    int *d_a, *d_b, *d_c;
    int size = N * N * sizeof(int);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N); // Launch N x N threads
    matrixMultiply<<<1, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
