
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x; // Thread index
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 5;
    int a[] = {1, 2, 3, 4, 5}, b[] = {10, 20, 30, 40, 50}, c[5];

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(int));
    hipMalloc(&d_b, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    vectorAdd<<<1, n>>>(d_a, d_b, d_c, n); // Launch kernel with n threads

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
