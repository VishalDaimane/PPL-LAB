
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printMessage(int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        printf("Welcome to Parallel Programming\n");
    }
}

int main() {
    int N, threadsPerBlock, blocksPerGrid;

    // Get user input for N (number of prints), threads per block, and blocks per grid
    printf("Enter the number of times to print the message: ");
    scanf("%d", &N);
    printf("Enter the number of threads per block: ");
    scanf("%d", &threadsPerBlock);
    printf("Enter the number of blocks per grid: ");
    scanf("%d", &blocksPerGrid);

    // Launch the kernel
    printMessage<<<blocksPerGrid, threadsPerBlock>>>(N);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    return 0;
}
