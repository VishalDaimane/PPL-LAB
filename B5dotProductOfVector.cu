
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5 // Size of the vectors

__global__ void dotProduct(int *a, int *b, int *result, int n) {
    int idx = threadIdx.x;
    int sum = 0;
    if (idx < n) {
        sum = a[idx] * b[idx];
    }
    atomicAdd(result, sum); // Add the result atomically
}

int main() {
    int a[N] = {1, 2, 3, 4, 5};
    int b[N] = {5, 4, 3, 2, 1};
    int result = 0;

    int *d_a, *d_b, *d_result;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_result, sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);

    dotProduct<<<1, N>>>(d_a, d_b, d_result, N);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Dot Product: %d\n", result);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
