
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function
__global__ void addNumbers(int a, int b, int *result) {
    *result = a + b;
}

int main() {
    int a = 5, b = 7, result;     // Host variables
    int *d_result;                // Device result

    hipMalloc(&d_result, sizeof(int));                    // Allocate memory on GPU
    addNumbers<<<1, 1>>>(a, b, d_result);                  // Launch kernel
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost); // Copy result back

    printf("Sum: %d\n", result);                           // Output result
    hipFree(d_result);                                    // Free GPU memory
    return 0;
}
